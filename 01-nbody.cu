#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"
#include "files.h"

#define SOFTENING 1e-9f

/*
 * Each body contains x, y, and z coordinate positions,
 * as well as velocities in the x, y, and z directions.
 */

typedef struct { float x, y, z, vx, vy, vz; } Body;

/*
 * Calculate the gravitational impact of all bodies in the system
 * on all others.
 */
__global__
void bodyForce(Body *p, float dt, int n) {
  int stride = blockDim.x * gridDim.x;
  for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < n; i += stride) {
    float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f;

    for (int j = 0; j < n; j++) {
        float dx = p[j].x - p[i].x;
        float dy = p[j].y - p[i].y;
        float dz = p[j].z - p[i].z;
        float distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
        float invDist = rsqrtf(distSqr);
        float invDist3 = invDist * invDist * invDist;
    
        Fx += dx * invDist3; Fy += dy * invDist3; Fz += dz * invDist3;
    }

    p[i].vx += dt*Fx; p[i].vy += dt*Fy; p[i].vz += dt*Fz;
  }
}


int main(const int argc, const char** argv) {
  int nBodies = 2<<15;
  if (argc > 1) nBodies = 2<<atoi(argv[1]);

  const char * initialized_values;
  const char * solution_values;

  if (nBodies == 2<<11) {
    initialized_values = "09-nbody/files/initialized_4096";
    solution_values = "09-nbody/files/solution_4096";
  } else { // nBodies == 2<<15
    initialized_values = "09-nbody/files/initialized_65536";
    solution_values = "09-nbody/files/solution_65536";
  }

  if (argc > 2) initialized_values = argv[2];
  if (argc > 3) solution_values = argv[3];

  const float dt = 0.01f; // Time step
  const int nIters = 10;  // Simulation iterations

  int bytes = nBodies * sizeof(Body);
  float *buf;

  hipMallocManaged(&buf, bytes);
  Body *p = (Body*)buf;

  read_values_from_file(initialized_values, buf, bytes);

  double totalTime = 0.0;

  int deviceId;
  hipGetDevice(&deviceId);
  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, deviceId);
  int multiProcessorCount = props.multiProcessorCount;
  int warpSize = props.warpSize;

  int blocks = multiProcessorCount * 32;
  int threads_per_block = warpSize * 8;

  for (int iter = 0; iter < nIters; iter++) {
    StartTimer();

    hipMemPrefetchAsync(p, bytes, deviceId);
    bodyForce<<<blocks, threads_per_block>>>(p, dt, nBodies);

    hipDeviceSynchronize();
  /*
   * This position integration cannot occur until this round of `bodyForce` has completed.
   * Also, the next round of `bodyForce` cannot begin until the integration is complete.
   */

    hipMemPrefetchAsync(p, bytes, hipCpuDeviceId);

    for (int i = 0 ; i < nBodies; i++) { // integrate position
      p[i].x += p[i].vx*dt;
      p[i].y += p[i].vy*dt;
      p[i].z += p[i].vz*dt;
    }

    const double tElapsed = GetTimer() / 1000.0;
    totalTime += tElapsed;
  }

  double avgTime = totalTime / (double)(nIters);
  float billionsOfOpsPerSecond = 1e-9 * nBodies * nBodies / avgTime;
  write_values_to_file(solution_values, buf, bytes);

  // You will likely enjoy watching this value grow as you accelerate the application,
  // but beware that a failure to correctly synchronize the device might result in
  // unrealistically high values.
  printf("%0.3f Billion Interactions / second\n", billionsOfOpsPerSecond);

  hipFree(buf);
}
